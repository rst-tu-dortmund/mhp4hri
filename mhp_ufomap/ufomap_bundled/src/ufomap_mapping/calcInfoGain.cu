#include "hip/hip_runtime.h"
/*********************************************************************
 *
 *  Software License Agreement
 *
 *  Copyright (c) 2023,
 *  TU Dortmund - Institute of Control Theory and Systems Engineering.
 *  All rights reserved.
 *
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with this program.  If not, see <https://www.gnu.org/licenses/>.
 *
 *  Authors: Heiko Renz
 *********************************************************************/

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <ufo/map/occupancy_map.h>
#include <ufo/map/occupancy_map_color.h>
#include <Eigen/Geometry>
#include <map>
#include <tuple>
// Macro source: https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{

    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Node definitions from Ufomap source
using INNER_NODE = ufo::map::OccupancyMapInnerNode<ufo::map::ColorOccupancyNode<float>>;
using LEAF_NODE  = ufo::map::OccupancyMapLeafNode<ufo::map::ColorOccupancyNode<float>>;
using DepthType  = unsigned int;

// Global function means it will be executed on the device (GPU)
// Calculates the information gain for rays to various startpoints
__global__ void calcInfoGain(double* out, ufo::map::OccupancyMapColor* map, ufo::map::Point3* origin, ufo::map::Point3* endpoints, double* max_range,
                             int* num_endpoints, int* num_startpoints)
{
    // Get the index of the current thread
    int index_x = threadIdx.x + blockIdx.x * blockDim.x;
    int index_y = threadIdx.y + blockIdx.y * blockDim.y;
    if ((index_x < num_endpoints[0]) && (index_y < num_startpoints[0]))  // if index outside of range, do nothing to avoid memory access errors
    {
        int index = index_x + index_y * num_endpoints[0];

        // Check if the endpoint and startpoint is within the map
        double min_allowed = map->getMin()[0];
        double max_allowed = map->getMax()[0];

        assert((min_allowed <= origin[blockIdx.y].min() && max_allowed >= origin[blockIdx.y].max() && min_allowed <= endpoints[index].min() &&
                max_allowed >= endpoints[index].max()));

        // Calculate the direction and distance of the ray
        ufo::map::Point3 direction = (endpoints[index] - origin[blockIdx.y]);
        double distance            = direction.norm();
        direction /= distance;

        // Check if the ray is within the max range
        if (0 <= max_range[0] && distance > max_range[0])
        {
            endpoints[index] = origin[blockIdx.y] + (direction * max_range[0]);
            distance         = max_range[0];
        }

        // Define variables for the raycasting
        ufo::map::Key current;
        ufo::map::Key ending;
        std::array<int, 3> step;
        ufo::map::Point3 t_delta;
        ufo::map::Point3 t_max;
        double info_val = 0;

        // Compute the raycasting
        map->computeRayInit(origin[blockIdx.y], endpoints[index], direction, current, ending, step, t_delta, t_max);
        // Increment
        int i         = 0;
        bool occupied = false;
        while ((current.getDepth() != ending.getDepth() || !current.equals(ending)) && t_max.min() <= distance && !occupied)
        {
            i++;
            // Compute the information gain for the current node
            ufo::map::OccupancyState state = map->getState(current);
            switch (state)
            {
                case ufo::map::OccupancyState::occupied:  // If the node is occupied, we
                                                          // want to know less about it
                                                          // --> Onyl small increae
                    info_val = info_val + (1 - map->getOccupancy(current));
                    occupied = true;
                    break;
                case ufo::map::OccupancyState::free:  // If the node is free,  we
                                                      // want to know less about it
                                                      // --> Onyl small increae
                    info_val = info_val + (map->getOccupancy(current));

                    break;
                case ufo::map::OccupancyState::unknown:  // If the node is unknown -->
                                                         // We want to know more
                    info_val++;
                    break;
            }
            // Increment the ray
            map->computeRayTakeStep(current, step, t_delta, t_max);
        }
        // Write the information gain to the output array
        out[index] = info_val / i;
    }
}

// This function is used to get the children pointer for all nodes from the host to the device
void getChildrenPointer(std::vector<void*>* children_h, INNER_NODE* root, ufo::map::OccupancyMapColor* map_h, int depth_cnt = 0, int depth_max = 16)
{

    // Initialize variables
    void* d_children;
    INNER_NODE& inner_node = *root;

    // Recursion stop condition if we reached the max depth
    if (depth_cnt == depth_max - 1) return;

    // Check if the current node is a leaf node and return
    if (!map_h->hasChildren(inner_node))
    {
        return;
    }
    else  // If the current node is an inner node, we want to get the children pointer
    {
        // Get the children pointer from the host (recursive call)
        for (int i = 0; i <= 7; i++)
        {
            INNER_NODE& child_node = static_cast<INNER_NODE&>(map_h->getChild(inner_node, 1, i));
            if (!child_node.is_leaf && child_node.children != nullptr) getChildrenPointer(children_h, &child_node, map_h, depth_cnt + 1);
        }
        // Push the children pointer to the vector
        children_h->push_back(inner_node.children);
        // Allocate memory on the device and copy the children pointer to the device
        gpuErrchk(hipMalloc((void**)&d_children, sizeof(INNER_NODE) * 8));
        gpuErrchk(hipMemcpy(d_children, inner_node.children, sizeof(INNER_NODE) * 8, hipMemcpyHostToDevice));
        // Set the children pointer of the current node to the device pointer for copying the whole map with device pointers
        root->children = d_children;
        return;
    }
    // }
}

// This function is used to set the children pointer back to the host pointers
void setPointerBackToHost(std::vector<void*>* children_h, INNER_NODE* root, ufo::map::OccupancyMapColor* map_h, int depth_cnt = 0, int depth_max = 16)
{
    // Initialize variables
    void* h_children       = (void*)malloc(sizeof(INNER_NODE) * 8);
    INNER_NODE& inner_node = *root;

    // Recursion stop condition if we reached the max depth
    if (depth_cnt == depth_max - 1) return;

    // Check if the current node is a leaf node and return
    if (!map_h->hasChildren(inner_node))
    {
        return;
    }
    else  // If the current node is an inner node, we want to get the children pointer back to the host
    {
        // Get the children pointer from the device
        gpuErrchk(hipMemcpy(h_children, inner_node.children, sizeof(INNER_NODE) * 8, hipMemcpyDeviceToHost));
        // Set the children pointer of the current node to the host pointer
        root->children = h_children;
        // Get the children pointer from the device and set to host pointer (recursive call)
        for (int i = 0; i <= 7; i++)
        {
            INNER_NODE& child_node = static_cast<INNER_NODE&>(map_h->getChild(inner_node, 1, i));
            if (!child_node.is_leaf && child_node.children != nullptr) setPointerBackToHost(children_h, &child_node, map_h, depth_cnt + 1);
        }
    }
}

void calcInfoGainGPU(int num, int threads, ufo::map::OccupancyMapColor* map,
                     std::vector<std::tuple<int, ufo::map::Point3, Eigen::AngleAxisd>>* start_points,
                     std::map<int, std::vector<ufo::map::Point3>>* endpoints, double max_range, Eigen::MatrixXd* results,
                     Eigen::Vector4d* time_metrics)
{
    std::chrono::time_point t1 = std::chrono::steady_clock::now();

    // Initialize variables
    ufo::map::OccupancyMapColor map_h(*map);

    // host copy of output
    double* out;
    std::vector<ufo::map::Point3> endpoints_h(start_points->size() * num);
    std::vector<ufo::map::Point3> startpoints_h(start_points->size());
    for (int i = 0; i < start_points->size(); i++)
    {
        startpoints_h[i]                  = std::get<1>(start_points->at(i));
        std::vector<ufo::map::Point3>* ep = &endpoints->at(std::get<0>(start_points->at(i)));
        for (int j = 0; j < num; j++)
        {
            endpoints_h[i * num + j] = ep->at(j);
        }
    }
    int num_startpoints = startpoints_h.size();

    // device copies of inputs and output
    double* d_out;
    int *d_num_endpoints, *d_num_startpoints;
    double* d_max_range;
    ufo::map::OccupancyMapColor* d_map;
    ufo::map::Point3* d_endpoints;
    ufo::map::Point3* d_start;

    // Get the children pointer from the host to the device
    std::vector<void*> children_h;
    children_h.clear();
    int depth_max    = map_h.getTreeDepthLevels();
    INNER_NODE* node = &map_h.getRoot();
    getChildrenPointer(&children_h, node, &map_h, 0, depth_max);

    // Alloc space for device copies
    gpuErrchk(hipMalloc((void**)&d_out, sizeof(double) * num * num_startpoints));                  // one double for each endpoint
    gpuErrchk(hipMalloc((void**)&d_map, sizeof(ufo::map::OccupancyMapColor)));                     // alloc map space
    gpuErrchk(hipMalloc((void**)&d_endpoints, sizeof(ufo::map::Point3) * num * num_startpoints));  // alloc space for endpoints
    gpuErrchk(hipMalloc(
        (void**)&d_start,
        sizeof(ufo::map::Point3) *
            num_startpoints));  // alloc space for startpoint (only one); TODO(renz): Check if parallelization for startpoints is also possible
    gpuErrchk(hipMalloc((void**)&d_max_range, sizeof(double)));     // alloc space for max_range double
    gpuErrchk(hipMalloc((void**)&d_num_endpoints, sizeof(int)));    // alloc space for num_endpoints int
    gpuErrchk(hipMalloc((void**)&d_num_startpoints, sizeof(int)));  // alloc space for num_startpoints int

    // Alloc space for host copies of info gain for all endpoints
    out = (double*)malloc(sizeof(double) * num * num_startpoints);  // one double for each endpoint

    // Copy data to device (Note that the node pointers for children are already copied to the device)
    gpuErrchk(hipMemcpy(d_map, &map_h, sizeof(ufo::map::OccupancyMapColor),
                         hipMemcpyHostToDevice));  // copy map to device (with new device pointers for children)

    gpuErrchk(hipMemcpy(d_max_range, &max_range, sizeof(double), hipMemcpyHostToDevice));           // copy max_range to device
    gpuErrchk(hipMemcpy(d_num_endpoints, &num, sizeof(int), hipMemcpyHostToDevice));                // copy num_endpoints to device
    gpuErrchk(hipMemcpy(d_num_startpoints, &num_startpoints, sizeof(int), hipMemcpyHostToDevice));  // copy num_endpoints to device
 

    gpuErrchk(hipMemcpy(d_start, startpoints_h.data(), sizeof(ufo::map::Point3) * num_startpoints,
                         hipMemcpyHostToDevice));  // copy startpoint to device
    gpuErrchk(hipMemcpy(d_endpoints, endpoints_h.data(), sizeof(ufo::map::Point3) * num * num_startpoints,
                         hipMemcpyHostToDevice));  // copy endpoints to device
    gpuErrchk(hipPeekAtLastError());  // Error check

    // Launch kernel
    dim3 gridDim(static_cast<int>(std::ceil(static_cast<double>(num) / static_cast<double>(threads))), num_startpoints);

    std::chrono::time_point t2 = std::chrono::steady_clock::now();

    calcInfoGain<<<gridDim, threads>>>(d_out, d_map, d_start, d_endpoints, d_max_range, d_num_endpoints, d_num_startpoints);
    gpuErrchk(hipDeviceSynchronize());

    std::chrono::time_point t3 = std::chrono::steady_clock::now();

    // Copy result back to host
    gpuErrchk(hipMemcpy(out, d_out, sizeof(double) * num * num_startpoints, hipMemcpyDeviceToHost));

    for (int blockIdx = 0; blockIdx < num_startpoints; blockIdx++)
    {
        results->row(blockIdx) = Eigen::Map<Eigen::VectorXd>(out + blockIdx * num, num);
    }

    // Copy result back to host
    setPointerBackToHost(&children_h, node, &map_h);  // Required since the destructor needs the correct host pointers to free the memory on the host

    free(out);

    // Cleanup on device
    hipFree(d_out);
    hipFree(d_map);
    hipFree(d_endpoints);
    hipFree(d_start);
    hipFree(d_max_range);
    hipFree(d_num_endpoints);
    hipFree(d_num_startpoints);

    std::chrono::time_point t4 = std::chrono::steady_clock::now();

    *time_metrics = Eigen::Vector4d{std::chrono::duration<float, std::chrono::seconds::period>(t2 - t1).count(),
                                     std::chrono::duration<float, std::chrono::seconds::period>(t3 - t2).count(),
                                     std::chrono::duration<float, std::chrono::seconds::period>(t4 - t3).count(),
                                     std::chrono::duration<float, std::chrono::seconds::period>(t2 - t1).count() +
                                         std::chrono::duration<float, std::chrono::seconds::period>(t4 - t3).count()};

}
